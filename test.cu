

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 5
__global__ void MatAdd(float d_A[N][N], float d_B[N][N], float d_C[N][N])
{
  int i = threadIdx.x;
  int j = threadIdx.y;
  if (i < N && j < N){
    d_C[i][j] = d_A[i][j] + d_B[i][j];
  }
  
}

__global__ void setElement(float d_A[N][N], float d_B[N][N], float d_C[N][N])
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < N && j < N){
  	d_A[i][j] = i * 3.2 + j * 2.21;
  	d_B[i][j] = i * 1.3 + j * 3.1;
  }
}


int main()
{
  int ARRAY_SIZE = N * N;

  int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

/*  int h_l,h_m,h_n,h_k;
  int d_l,d_m,d_n,d_k;
*/
/*  h_m = atoi((const char *)argv[1]);
  h_n = atoi((const char *)argv[2]);
  h_k = atoi((const char *)argv[3]);*/

	//float h_C[N][N];

  float h_A[N][N], h_B[N][N], h_C[N][N];
  float (*d_A)[N], (*d_B)[N], (*d_C)[N];

  hipMalloc(&d_A, ARRAY_BYTES);
  hipMalloc((void**) &d_B, ARRAY_BYTES);
  hipMalloc((void**) &d_C, ARRAY_BYTES);
  
  // Kernel invocation with one block of N * N * 1 threads
  int numBlocks = 1;
  dim3 threadsPerBlock(N, N);
  setElement<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);

  hipMemcpy(h_A, d_A, ARRAY_BYTES, hipMemcpyDeviceToHost);
  hipMemcpy(h_B, d_B, ARRAY_BYTES, hipMemcpyDeviceToHost);
  
  for (int i=0; i < N; i++)
  {
	for (int j=0; j < N; j++)
	{
		printf("%f", h_A[i][j]);
		printf("  %f\n", h_B[i][j]);
	}
  }
  
  
  MatAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);

  hipMemcpy(h_C, d_C, ARRAY_BYTES, hipMemcpyDeviceToHost);
  for (int i=0; i < N; i++)
  {
  for (int j=0; j < N; j++)
  {
    printf("%f\n", h_C[i][j]);
  }
  }


}

//nvcc -o test test.cu 