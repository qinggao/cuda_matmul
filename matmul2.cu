#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <time.h>
#include "gputimer.h"

#define N 800
__global__ void MatMul(float d_A[N][N], float d_B[N][N], float d_C[N][N])
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;   
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  
  if (i < N && j < N)
  {
    for (int l = 0; l < N; l++)
    {
      //d_C[i][j] = d_C[i][j] + d_A[j][l] * d_B[l][i];
      d_C[i][j] = d_C[i][j] + d_A[i][l] * d_B[l][j];
    }
  }
}

__global__ void setElement(float d_A[N][N], float d_B[N][N], float d_C[N][N])
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < N && j < N){
  	d_A[i][j] = i * (float)3.2 + j * (float)2.21;
  	d_B[i][j] = i * (float)1.3 + j * (float)3.1;
    d_C[i][j] = (float)0;
  }
}

int main()
{
  GpuTimer timer;

  //int m,n,k;
  //m = n = k = N;

  //int i,j;
  int ARRAY_SIZE = N * N;

  int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);


  float h_A[N][N], h_B[N][N], h_C[N][N];
  float (*d_A)[N], (*d_B)[N], (*d_C)[N];

  hipMalloc((void**) &d_A, ARRAY_BYTES);
  hipMalloc((void**) &d_B, ARRAY_BYTES);
  hipMalloc((void**) &d_C, ARRAY_BYTES);
  
  // Kernel invocation with least amount of blocks
  //int numBlocks;
  int block_x = ceil((float)N / (float)512);
  int block_y = 1; //ceil((float)N / (float)22);

  dim3 numBlocks(block_x, block_y);
  

  //dim3 threadsPerBlock(22, 22);

  setElement<<<numBlocks, 512>>>(d_A, d_B, d_C)

    /*timer.Start();
  setElement<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);
  MatMul<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);
  timer.Stop();*/


  hipMemcpy(h_A, d_A, ARRAY_BYTES, hipMemcpyDeviceToHost);
  hipMemcpy(h_B, d_B, ARRAY_BYTES, hipMemcpyDeviceToHost);
  hipMemcpy(h_C, d_C, ARRAY_BYTES, hipMemcpyDeviceToHost);
  
  
/*    fprintf(stdout, "Here is the matrix A:\n\n");
  for(i=0;i<m;i++) {
    for(j=0;j<k;j++) {
      fprintf(stdout, "%10.2f",h_A[i][j]);
    }
    fprintf(stdout, "\n");
  }
  fprintf(stdout, "Here is the matrix B:\n\n");
  for(i=0;i<k;i++) {
    for(j=0;j<n;j++) {
      fprintf(stdout, "%10.2f",h_B[i][j]);
    }
    fprintf(stdout, "\n");
  }

    fprintf(stdout, "Here is the matrix C:\n\n");
  for(i=0;i<m;i++) {
    for(j=0;j<n;j++) {
      fprintf(stdout, "%10.2f",h_C[i][j]);
    }
    fprintf(stdout, "\n");
  }*/

  printf("Time elapsed = %g ms\n", timer.Elapsed());


  // Clean up memory
  hipHostFree(h_A);
  hipHostFree(h_B);
  hipHostFree(h_C);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);


}
