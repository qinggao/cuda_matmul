
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <time.h>
#include <sys/resource.h>

#define N 10
__global__ void MatMul(float d_A[N][N], float d_B[N][N], float d_C[N][N])
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;   
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  
  if (i < N && j < N)
  {
    for (int l = 0; l < N; l++)
    {
      d_C[i][j] = d_C[i][j] + d_A[j][l] * d_B[l][i];
    
    }
  }
}

__global__ void setElement(float d_A[N][N], float d_B[N][N], float d_C[N][N])
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < N && j < N){
  	d_A[i][j] = i * (float)3.2 + j * (float)2.21;
  	d_B[i][j] = i * (float)1.3 + j * (float)3.1;
    d_C[i][j] = (float)0;
  }
}

int main()
{

  int m,n,k;
  m = n = k = N;

  int i,j;
  int ARRAY_SIZE = N * N;

  int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);


  float h_A[N][N], h_B[N][N], h_C[N][N];
  float (*d_A)[N], (*d_B)[N], (*d_C)[N];

  hipMalloc((void**) &d_A, ARRAY_BYTES);
  hipMalloc((void**) &d_B, ARRAY_BYTES);
  hipMalloc((void**) &d_C, ARRAY_BYTES);
  
  // Kernel invocation with one block of N * N * 1 threads
  int numBlocks = 5;
  dim3 threadsPerBlock(N, N);
  setElement<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);

  hipMemcpy(h_A, d_A, ARRAY_BYTES, hipMemcpyDeviceToHost);
  hipMemcpy(h_B, d_B, ARRAY_BYTES, hipMemcpyDeviceToHost);
  hipMemcpy(h_C, d_C, ARRAY_BYTES, hipMemcpyDeviceToHost);
  
  
    fprintf(stdout, "Here is the matrix A:\n\n");
  for(i=0;i<m;i++) {
    for(j=0;j<k;j++) {
      fprintf(stdout, "%10.2f ",h_A[i][j]);
    }
    fprintf(stdout, "\n");
  }
  fprintf(stdout, "Here is the matrix B:\n\n");
  for(i=0;i<k;i++) {
    for(j=0;j<n;j++) {
      fprintf(stdout, "%10.2f",h_B[i][j]);
    }
    fprintf(stdout, "\n");
  }

  
  MatMul<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);

  hipMemcpy(h_C, d_C, ARRAY_BYTES, hipMemcpyDeviceToHost);


    fprintf(stdout, "Here is the matrix C:\n\n");
  for(i=0;i<m;i++) {
    for(j=0;j<n;j++) {
      fprintf(stdout, "%10.2f",h_C[i][j]);
    }
    fprintf(stdout, "\n");
  }


  // Clean up memory
  hipHostFree(h_A);
  //free(h_B);
  hipHostFree(h_C);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);


}

//nvcc -o test test.cu 