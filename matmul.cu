
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <time.h>
#include <sys/resource.h>

#define N 3
__global__ void MatMul(float d_A[N][N], float d_B[N][N], float d_C[N][N])
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;   
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  
  for (int l = 0; l < N; l++)
  {
    d_C[i][j] = d_C[i][j] + d_A[l][j] * d_B[i][l];
  
  }
  
  /*if (i < N && j < N){
    d_C[i][j] = d_A[i][j] + d_B[i][j];
  }*/
  
}

__global__ void setElement(float d_A[N][N], float d_B[N][N], float d_C[N][N])
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < N && j < N){
  	d_A[i][j] = i * (float)3.2 + j * (float)2.21;
  	d_B[i][j] = i * (float)1.3 + j * (float)3.1;
    d_C[i][j] = (float)0;
  }
}


int main()
{

  int m,n,k;
  m = n = k = N;

  int i,j;
  int ARRAY_SIZE = N * N;

  int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

/*  int h_l,h_m,h_n,h_k;
  int d_l,d_m,d_n,d_k;
*/
/*  h_m = atoi((const char *)argv[1]);
  h_n = atoi((const char *)argv[2]);
  h_k = atoi((const char *)argv[3]);*/

	//float h_C[N][N];

  float h_A[N][N], h_B[N][N], h_C[N][N];
  float (*d_A)[N], (*d_B)[N], (*d_C)[N];

  hipMalloc((void**) &d_A, ARRAY_BYTES);
  hipMalloc((void**) &d_B, ARRAY_BYTES);
  hipMalloc((void**) &d_C, ARRAY_BYTES);
  
  // Kernel invocation with one block of N * N * 1 threads
  int numBlocks = 5;
  dim3 threadsPerBlock(N, N);
  setElement<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);

  hipMemcpy(h_A, d_A, ARRAY_BYTES, hipMemcpyDeviceToHost);
  hipMemcpy(h_B, d_B, ARRAY_BYTES, hipMemcpyDeviceToHost);
  hipMemcpy(h_C, d_C, ARRAY_BYTES, hipMemcpyDeviceToHost);
  
/*  for (int i=0; i < N; i++)
  {
	for (int j=0; j < N; j++)
	{
		printf("%f", h_A[i][j]);
		printf("  %f\n", h_B[i][j]);
	}
  }*/
  
    fprintf(stdout, "Here is the matrix A:\n\n");
  for(i=0;i<m;i++) {
    for(j=0;j<k;j++) {
      fprintf(stdout, "%10.2f ",h_A[i][j]);
    }
    fprintf(stdout, "\n");
  }
  fprintf(stdout, "Here is the matrix B:\n\n");
  for(i=0;i<k;i++) {
    for(j=0;j<n;j++) {
      fprintf(stdout, "%10.2f",h_B[i][j]);
    }
    fprintf(stdout, "\n");
  }

  fprintf(stdout, "Here is the matrix B:\n\n");
  for(i=0;i<k;i++) {
    for(j=0;j<n;j++) {
      fprintf(stdout, "%10.2f",h_C[i][j]);
    }
    fprintf(stdout, "\n");
  }




  
  MatMul<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);

  hipMemcpy(h_C, d_C, ARRAY_BYTES, hipMemcpyDeviceToHost);


    fprintf(stdout, "Here is the matrix C:\n\n");
  for(i=0;i<m;i++) {
    for(j=0;j<n;j++) {
      fprintf(stdout, "%10.2f",h_C[i][j]);
    }
    fprintf(stdout, "\n");
  }

/*  for (int i=0; i < N; i++)
  {
  for (int j=0; j < N; j++)
  {
    printf("%f\n", h_C[i][j]);
  }
  }*/

  // Clean up memory
  hipHostFree(h_A);
  //free(h_B);
  hipHostFree(h_C);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);


}

//nvcc -o test test.cu 